#include "hip/hip_runtime.h"
#include "iostream"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipblas.h"
#include "pch.h"
//#include "stdafx.h"
#include "cv.h"
//#include <process.h>
//#include "CameraApi.h"
#include "LaserRange.h"
//#include "afxwin.h"
//#include "windows.h"
#include "math.h"
//#include "cstdlib"
//#include "sstream"
//#include "ImProcess.h"
//#include "opencv2/core/core.hpp"
//#include "opencv2/imgproc/imgproc.hpp"
//#include "opencv2/calib3d/calib3d.hpp"
#include "CudaTest.h"
#include <>
#include <hip/device_functions.h>

//#include <stdio.h>
//#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
//#include "CudaKernal.cuh"

using namespace std;
using namespace cv;
using namespace cv::cuda;

//extern "C" void GetGaussFitCuda(GpuMat gpuMat, MPoint *point, double maxError, double minError, int yRange, int Colonce);

//����˷� a[M][N]*b[N][S]
hipError_t checkCudaError(hipError_t CudaFunction, const char* ident) {
	hipError_t err = CudaFunction;
	if (err != hipSuccess) {
		fprintf(stderr, "%s \t hipError_t:%s\n", ident, hipGetErrorString(hipGetLastError()));
	}
	return err;
}

//Coloncel��ɨ��õ�洢
__global__ void GetGaussPointCuda(PtrStepSz<uchar1> src, MPoint *point, int **gpu_data, int Colonce, int Rows, int Cols) {
	int threadId = threadIdx.x;
	//printf("%d\n",threadId);
	//__shared__ int *gpu_cr;
	//gpu_cr = new int [Rows*Cols];
	//���д�������
	for (int j = 0; j < Rows; j++)
	{
		for (int i = 0; i < Colonce; i++)
		{
			gpu_data[i + threadId * Colonce][j] = (int)src(j, threadId*Colonce + i).x;
		}
	}
	//int i = 0, j = 0;
	//gpu_cr[i*Rows + j] = *((int*)&src( j, threadId*Colonce + i));

	//ȡÿ�����ֵλ��
	for (int i = 0; i < Colonce; i++) {
		int MaxPixel = gpu_data[i + threadId * Colonce][0];
		//printf("the first pixel is %d \n", MaxPixel);
		int MaxY = 0;
		for (int j = 1; j < Rows; j++)
		{
			if (gpu_data[i + threadId * Colonce][j] > MaxPixel)
			{
				MaxPixel = gpu_data[i + threadId * Colonce][j];
				MaxY = j;
			}
		}

		point[threadId*Colonce + i].x = threadId * Colonce + i;
		point[threadId*Colonce + i].y = MaxY;
		point[threadId*Colonce + i].bright = MaxPixel;
	}
	__syncthreads();
}

//����ɸѡ�������˹��
__global__ void GetGaussFitRes(MPoint *point, MatrixUnion *gpu_mar, int **gpu_data, double maxError, double minError, int yRange, int Rows, int Cols, int Precision)
{
	//ͨ���鲢�н��һ��block��thread�����õ�����
	int threadId = blockIdx.x*blockDim.x + threadIdx.x;
	//�ж���ȷ�����߳��пɴ�������
	if (threadId < Cols)
	{
		////��˹��洢����
		//int *y; //�洢��˹����ÿ�е���λ��
		//int *br; //�洢��˹���ֵ
		GPoint *gpoint = new GPoint[2 * yRange];
		int Pixnum = 0; //ͳ�Ƹ�˹�����
		//ȷ�����½�λ�� ���ټ������
		double minLine = minError * point[threadId].bright;
		double maxLine = (1 - maxError) * point[threadId].bright;
		//��˹��ɸѡ
		for (int i = (point[threadId].y - yRange); i < (point[threadId].y + yRange + 1); i++)
		{
			if ((gpu_data[threadId][i] > minLine) && (gpu_data[threadId][i] < maxLine))
			{
				gpoint[Pixnum].x = i;
				gpoint[Pixnum].brightness = gpu_data[threadId][i];
				Pixnum++;
			}
		}
		point[threadId].Pixnum = Pixnum;
		//__shared__ MatrixUnion *gpu_mar;

		//��˹�����3ʱ�������
		if (Pixnum > 3) {
			//�����������
			int n = Pixnum;

			//X����1 x x^2��  n*3
			gpu_mar[threadId].X = new long long int *[n];
			for (int i = 0; i < n; i++) {
				gpu_mar[threadId].X[i] = new long long int[3];
			}
			//XT���� X��ת��  3*n
			gpu_mar[threadId].XT = new long long int *[3];
			for (int i = 0; i < 3; i++) {
				gpu_mar[threadId].XT[i] = new long long int[n];
			}
			//Z����(brightness) n*1
			gpu_mar[threadId].Z = new long long int[n];
			//B���󣨽���� 3*1
			gpu_mar[threadId].B = new long long int[3];
			//SA���� ��XT*X�� 3*3
			gpu_mar[threadId].SA = new long long int *[3];
			for (int i = 0; i < 3; i++) {
				gpu_mar[threadId].SA[i] = new long long int[3];
			}
			//SAN���� SA�������  3*3
			gpu_mar[threadId].SAN = new float *[3];
			for (int i = 0; i < 3; i++) {
				gpu_mar[threadId].SAN[i] = new float[3];
			}
			//SC����  SAN*XT 3*n
			gpu_mar[threadId].SC = new float *[3];
			for (int i = 0; i < 3; i++) {
				gpu_mar[threadId].SC[i] = new float[n];
			}


			/*
			//X����1 x x^2��  n*3
			double **X = new double*[n];
			for (int i = 0; i < n; i++) {
				X[i] = new double[3];
			}

			//XT���� X��ת��  3*n
			double **XT = new double*[3];
			for (int i = 0; i < 3; i++) {
				XT[i] = new double[n];
			}

			//Z����(brightness) n*1
			double *Z = new double[n];
			//B���󣨽���� 3*1
			double *B = new double[3];

			//SA���� ��XT*X�� 3*3
			double **SA = new double*[3];
			for (int i = 0; i < 3; i++) {
				SA[i] = new double[3];
			}

			//SAN���� SA�������  3*3
			double **SAN = new double*[3];
			for (int i = 0; i < 3; i++) {
				SAN[i] = new double[3];
			}
			*/

			//����X�����Z���� ˳�ִ���ת��XT
			for (int i = 0; i < n; i++) {
				gpu_mar[threadId].X[i][0] = 1 * Precision;
				gpu_mar[threadId].X[i][1] = gpoint[i].x*Precision;
				gpu_mar[threadId].X[i][2] = gpoint[i].x*gpoint[i].x*Precision;
				gpu_mar[threadId].Z[i] = gpoint[i].brightness*Precision;
				gpu_mar[threadId].XT[0][i] = 1 * Precision;
				gpu_mar[threadId].XT[1][i] = gpoint[i].x*Precision;
				gpu_mar[threadId].XT[2][i] = gpoint[i].x*gpoint[i].x*Precision;
			}
			/*for (int i_ = 0; i_ < n; i_++)
			{
				for (int j_ = 0; j_ < 3; j_++)
				{
					printf("%d\t", gpu_mar[threadId].X[i_][j_]);
				}
				printf("\n");

			}*/
			//����SA = XT*X
			for (int m = 0; m < 3; m++) {
				for (int s = 0; s < 3; s++) {
					gpu_mar[threadId].SA[m][s] = 0;
					for (int i = 0; i < n; i++) {
						gpu_mar[threadId].SA[m][s] += gpu_mar[threadId].XT[m][i] * gpu_mar[threadId].X[i][s];
					}
				}
			}

			/*if (threadId == 20) {
				for (int i_ = 0; i_ < 3; i_++)
				{
					for (int j_ = 0; j_ < 3; j_++)
					{
						printf("%lld\t", gpu_mar[threadId].SA[i_][j_]);
					}
					printf("\n");
				}
			}*/

			//����SAN
			const int mat_num = 3;//�������Ľ���
			gpu_mar[threadId].in_v = new float*[mat_num];
			float **in_v = gpu_mar[threadId].in_v;
			for (int i_ = 0; i_ < mat_num; i_++)
			{
				in_v[i_] = new float[mat_num];
			}
			gpu_mar[threadId].BC = new float*[mat_num];
			float **BC = gpu_mar[threadId].BC;

			for (int i_ = 0; i_ < mat_num; i_++)
			{
				BC[i_] = new float[mat_num];
			}

			for (int i_ = 0; i_ < mat_num; i_++)
			{
				BC[i_][i_] = 1;
				for (int j_ = 0; j_ < mat_num; j_++)
				{
					if (j_ == i_)
						continue;
					BC[i_][j_] = 0;
				}
			}

			for (int i = 0; i < mat_num; i++)
			{
				//L������
				gpu_mar[threadId].L = new float*[mat_num];
				float **L = gpu_mar[threadId].L;
				for (int i_ = 0; i_ < mat_num; i_++)
				{
					L[i_] = new float[mat_num];
				}
				//U������
				gpu_mar[threadId].U = new float*[mat_num];
				float **U = gpu_mar[threadId].U;
				for (int i_ = 0; i_ < mat_num; i_++)
				{
					U[i_] = new float[mat_num];
				}

				gpu_mar[threadId].P = new int[mat_num];
				int *P = gpu_mar[threadId].P;
				//SA����
				gpu_mar[threadId].A_mirror = new long long int*[mat_num];
				long long int **A_mirror = gpu_mar[threadId].A_mirror;
				for (int i_ = 0; i_ < mat_num; i_++)
				{
					A_mirror[i_] = new long long int[mat_num];
				}
				for (int i_ = 0; i_ < mat_num; i_++)
				{
					for (int j_ = 0; j_ < mat_num; j_++)
					{
						A_mirror[i_][j_] = gpu_mar[threadId].SA[i_][j_];
					}
				}


				//����part  ��û�п���
				int  row = 0;
				for (int i_ = 0; i_ < mat_num; i_++)
					P[i_] = i_;
				for (int i_ = 0; i_ < mat_num - 1; i_++)
				{
					int p = 0;
					for (int j_ = i_; j_ < mat_num; j_++)
					{
						if (A_mirror[j_][i_] > p || (-1)* A_mirror[j_][i_] > p)
						{
							p = A_mirror[j_][i_] > 0 ? A_mirror[j_][i_] : (-1)*A_mirror[j_][i_];
							row = j_;
						}
					}

					/*
					if (p == 0)
					{
						;
					}
					*/

					//����P[i_] P[row]
					int tmp = P[i_];
					P[i_] = P[row];
					P[row] = tmp;

					//����A[i_][j_] ��A[row][j_]
					float tmp2 = 0;
					for (int j_ = 0; j_ < mat_num; j_++)
					{
						tmp2 = A_mirror[i_][j_];
						A_mirror[i_][j_] = A_mirror[row][j_];
						A_mirror[row][j_] = tmp2;
					}

					float u = A_mirror[i_][i_], l = 0;
					for (int j_ = i_ + 1; j_ < mat_num; j_++)
					{
						l = A_mirror[j_][i_] / u;
						A_mirror[j_][i_] = l;
						for (int k_ = i_ + 1; k_ < mat_num; k_++)
						{
							A_mirror[j_][k_] = A_mirror[j_][k_] - A_mirror[i_][k_] * l;
						}

					}

				}

				//����LU
				for (int i_ = 0; i_ < mat_num; i_++)
				{
					for (int j_ = 0; j_ <= i_; j_++)
					{
						if (i_ != j_)
							L[i_][j_] = A_mirror[i_][j_];
						else
							L[i_][j_] = 1;
					}
					for (int k_ = i_; k_ < mat_num; k_++)
					{
						U[i_][k_] = A_mirror[i_][k_];
					}
				}

				float *y = new float[mat_num];
				//�����滻
				for (int i_ = 0; i_ < mat_num; i_++)
				{
					y[i_] = BC[i][P[i_]];
					for (int j_ = 0; j_ < i_; j_++)
					{
						y[i_] = y[i_] - L[i_][j_] * y[j_];
					}
				}
				//�����滻
				for (int i_ = mat_num - 1; i_ >= 0; i_--)
				{
					in_v[i][i_] = y[i_];
					for (int j_ = mat_num - 1; j_ > i_; j_--)
					{
						in_v[i][i_] = in_v[i][i_] - U[i_][j_] * in_v[i][j_];
					}
					in_v[i][i_] /= U[i_][i_];
				}


				for (int i_ = 0; i_ < mat_num; i_++)
					delete[] L[i_];
				delete[]L;
				for (int i_ = 0; i_ < mat_num; i_++)
					delete[]U[i_];
				delete[]U;
				for (int i_ = 0; i_ < mat_num; i_++)
					delete[]A_mirror[i_];
				delete[]A_mirror;

				delete[]P;
				delete[]y;


			}

			for (int i_ = 0; i_ < mat_num; i_++)
			{
				for (int j_ = 0; j_ < i_; j_++)
				{
					gpu_mar[threadId].SAN[i_][j_] = in_v[j_][i_];
					gpu_mar[threadId].SAN[j_][i_] = in_v[i_][j_];
				}
				gpu_mar[threadId].SAN[i_][i_] = in_v[i_][i_];
			}

			for (int i = 0; i < mat_num; i++)
				delete[]in_v[i];
			delete[]in_v;
			for (int i = 0; i < mat_num; i++)
				delete[]BC[i];
			delete[]BC;
			/*
						if (threadId == 20) {
							for (int i_ = 0; i_ < mat_num; i_++)
							{
								for (int j_ = 0; j_ < mat_num; j_++)
								{
									printf("%lf\t", gpu_mar[threadId].SAN[i_][j_]);
								}
								printf("\n");
							}
						}*/

						//����SC = SAN*XT
			for (int m = 0; m < 3; m++) {
				for (int s = 0; s < n; s++) {
					gpu_mar[threadId].SC[m][s] = 0;
					for (int i = 0; i < 3; i++) {
						gpu_mar[threadId].SC[m][s] += gpu_mar[threadId].SAN[m][i] * gpu_mar[threadId].XT[i][s];
					}
				}
			}
			/*if (threadId == 20) {
				for (int i_ = 0; i_ < 3; i_++)
				{
					for (int j_ = 0; j_ < n; j_++)
					{
						printf("%lf\t", gpu_mar[threadId].SC[i_][j_]);
					}
					printf("\n");
				}
			}*/
			//����B = SC*Z
			for (int m = 0; m < 3; m++) {
				gpu_mar[threadId].B[m] = 0;
				for (int i = 0; i < n; i++) {
					gpu_mar[threadId].B[m] += gpu_mar[threadId].SC[m][i] * gpu_mar[threadId].Z[i];
				}
			}
			//����B
			//printf("B1 = %d , B2 = %d\n", gpu_mar[threadId].B[1], gpu_mar[threadId].B[2]);
			point[threadId].cx = threadId;
			point[threadId].cy = 1.0*(-gpu_mar[threadId].B[1]) / ((2 * gpu_mar[threadId].B[2]));

			//point[threadId].gaussbright = exp((float)(gpu_mar[threadId].B[0] - gpu_mar[threadId].B[1] * gpu_mar[threadId].B[1] / (4 * gpu_mar[threadId].B[2])));


			for (int i = 0; i < n; i++) {
				delete[] gpu_mar[threadId].X[i];
			}
			delete[] gpu_mar[threadId].X;
			for (int i = 0; i < 3; i++) {
				delete[] gpu_mar[threadId].XT[i];
			}
			delete[] gpu_mar[threadId].XT;
			delete[] gpu_mar[threadId].Z;
			delete[] gpu_mar[threadId].B;
			for (int i = 0; i < 3; i++) {
				delete[] gpu_mar[threadId].SA[i];
			}
			delete[] gpu_mar[threadId].SA;
			for (int i = 0; i < 3; i++) {
				delete[] gpu_mar[threadId].SAN[i];
			}
			delete[] gpu_mar[threadId].SAN;
			for (int i = 0; i < 3; i++) {
				delete[] gpu_mar[threadId].SC[i];
			}
			delete[] gpu_mar[threadId].SC;

		}
		else
		{
			point[threadId].cx = threadId;
			point[threadId].cy = 0;
			point[threadId].bright = 0;
		}



		delete[] gpoint;

	}


}

//��˹��ɸѡ
//for (int i = 0; i < Colonce; i++)
//{
//	int Pixnum = 0;
//	//GPoint *gpoint;
//	//point[threadId*Colonce+i].gpoint = new GPoint[Rows];
//	//point[i].gpoint = new GPoint[Rows];
//	for (int j = 0; j < Rows; j++)
//	{
//		if ((gpu_cr[Rows*i + j] > minError*point[threadId*Colonce + i].bright)
//			&& (gpu_cr[Rows*i + j] < (1 - maxError)*point[threadId*Colonce + i].bright)
//			&& (abs(j - point[threadId*Colonce + i].y) < yRange))
//		{
//			point[threadId*Colonce + i].gpoint[Pixnum].x = threadId * Colonce + i;
//			point[threadId*Colonce + i].gpoint[Pixnum].brightness = gpu_cr[Rows*i + j];
//			Pixnum++;
//		}
//		if ((j - point[threadId*Colonce + i].y) < yRange)
//			break;
//	}
//	point[threadId*Colonce + i].Pixnum = Pixnum;

	/*
	//��������
	if (Pixnum >= 3)
	{
		__shared__ int *X;
		X = new int[Pixnum * 3];
		__shared__ int *Z;
		Z = new int[Pixnum];
		//��������<3.5 ����Ƕ�ײ��к˺���
		//dim3 blockSEX(1, 0, 0);
		//dim3 threadSEX(Pixnum, 0, 0);
		//����X��Z����
		//SetElementX << <blockSEX, threadSEX >> > (gpoint, X, Pixnum);
		//����X����(n*3) Z����n*1)
		for (int i = 0; i < Pixnum; i++)
		{
			for (int j = 0; j < 3; j++)
			{
				if (j = 0)
				{
					X[i * 3 + j] = 1;
				}
				if (j = 1)
				{
					X[i * 3 + j] = gpoint[i].x;
				}
				if (j = 2)
				{
					X[i * 3 + j] = gpoint[i].x*gpoint[i].x;
				}
			}
			Z[i] = gpoint[i].brightness;
		}
		//��Xת��
		__shared__ int *XT;
		XT = new int[Pixnum* 3];
		for (int i = 0; i < 3; i++)
		{
			for (int j = 0; j < Pixnum; j++)
			{
				XT[i*Pixnum + j] = X[j * 3 + i];
			}
		}
		//��XT*X���
		__shared__ int *SA;
		SA = new int[3 * 3];
		for (int m = 0; i < 3; i++)
		{
			for (int s = 0; s < 3; s++)
			{
				for (int n = 0; n < Pixnum; n++)
				{
					SA[m * 3 + s] = XT[m*Pixnum + n] * X[n * 3 + s];
				}
			}
		}
		//��SA�����
		__shared__ int *SAN;
		SAN = new int[3 * 3];


	}*/
	//}
	//delete &gpu_cr;


//#define N 3
//__global__ void MatAdd(const int **A, const int **B, int **C)
//{
//	int i = threadIdx.x;
//	int j = threadIdx.y;
//	C[i][j] = A[i][j] + B[i][j];
//	//__syncthreads();
//}



//extern "C" void GetGaussFitCuda(GpuMat gpuMat, MPoint *point, double maxError, double minError, int yRange, int Colonce);

extern "C"
void CudaGuassHC(Mat matImage, MPoint *point, double maxError, double minError, int yRange, int Colonce, int Precision) {

	int Rows = matImage.rows;
	int Cols = matImage.cols;// *matImage.channels();
	//InputArray inputMat(matImage);
	//for (int j = 0; j < Rows; j++) {
	//	//uchar* data = gpuMat.ptr<uchar>(j);
	//	for (int i = 0; i < Cols; i++) {
	//		int datt = inputMat.ptr<uchar>(j)[i];
	//		//cout << "(" << i << "," <<j << "):" << datt << endl;
	//		printf("(%d,%d):%d\n", i, j, datt);
	//	}
	//}
	//cout << Cols << endl;
	GpuMat gpuMat(matImage);
	//gpuMat.upload(matImage);
	//for (int j = 0; j < Rows; j++) {
	//	//uchar* data = gpuMat.ptr<uchar>(j);
	//	for (int i = 0; i < Cols; i++) {
	//		int datt = gpuMat.ptr<uchar>(j)[i];
	//		//cout << "(" << i << "," <<j << "):" << datt << endl;
	//		printf("(%d,%d):%d\n", i, j, datt);
	//	}
	//}
	//�ṹ��ָ���ϴ�
	MPoint *gpu_point;
	//gpu_point = new MPoint[Cols];	
	checkCudaError(hipMalloc((void**)&gpu_point, sizeof(MPoint)*Cols), "malloc error1");
	//�Դ�ͼ�񻺴����
	int **gpu_data;
	int *gpu_data_d;
	int **cpu_data = (int**)malloc(sizeof(int*)*Cols);
	int *cpu_data_d = (int*)malloc(sizeof(int)*Cols*Rows);
	checkCudaError(hipMalloc((void**)&gpu_data, Cols * sizeof(int**)), "malloc error2");
	checkCudaError(hipMalloc((void**)&gpu_data_d, Cols *Rows * sizeof(int)), " malloc error2");
	for (int i = 0; i < Cols; i++) {
		cpu_data[i] = gpu_data_d + Rows * i;
		//�׵�ַ��ֵ ��һά����תΪ��ά
	}
	checkCudaError(hipMemcpy(gpu_data, cpu_data, sizeof(int*)*Cols, hipMemcpyHostToDevice), "memcpy error1");
	checkCudaError(hipMemcpy(gpu_data_d, cpu_data_d, sizeof(int)*Rows*Cols, hipMemcpyHostToDevice), "memcpy error1");

	/*if (hipSuccess != hipMemcpy(gpu_point, point, sizeof(point)*Cols, hipMemcpyHostToDevice)) {
		printf("cuda memcpy up error1!\n");
	}*/

	//dim3 threads_all(Cols / Colonce);
	//GPU�˾���
	MatrixUnion *gpu_mar;
	checkCudaError(hipMalloc((void**)&gpu_mar, sizeof(MatrixUnion)*Cols), "malloc error3");
	//ÿcolonce��ͳһ���� 
	GetGaussPointCuda << <1, Cols / Colonce >> > (gpuMat, gpu_point, gpu_data, Colonce, Rows, Cols);
	hipDeviceSynchronize();
	//�滮������  ֮�����Ϊֻ�滮һ��
	const int BlockPMat = 1280;
	int Blocknum, Threadnum;
	Blocknum = BlockPMat;
	Threadnum = Cols / Blocknum + 1;
	/*
	if (Cols > 1024) {
		Blocknum = Cols / 1024 + 1;
		Threadnum = 1024;
	}
	else {
		Blocknum = 1;
		Threadnum = Cols;
	}
	*/
	//���и�˹���
	GetGaussFitRes << <Blocknum, Threadnum >> > (gpu_point, gpu_mar, gpu_data, maxError, minError, yRange, Rows, Cols, Precision);
	hipDeviceSynchronize();
	checkCudaError(hipMemcpy(point, gpu_point, sizeof(MPoint)*Cols, hipMemcpyDeviceToHost), "memcpy down error1");

	for (int i = 0; i < Cols; i++)
	{
		//cout << "("<<point[i].x<<","<< point[i].y<<"):"<< point[i].bright << endl;
		printf("(%d,%d):%d\t, here are %d gausspoints, the result is ��%lf,%lf��\n", point[i].x, point[i].y, point[i].bright, point[i].Pixnum, point[i].cx, point[i].cy);
	}


	/*for (int i = 0; i < Cols; i++)
	{
		free((void*)cpu_data[i]);
	}*/
	free((void*)cpu_data);
	free(cpu_data_d);
	/*for (int i = 0; i < Cols; i++)
	{
		hipFree((void*)gpu_data[i]);
	}*/
	hipFree(gpu_data);
	hipFree(gpu_point);
	hipFree(gpu_data_d);
	hipFree(gpu_mar);
	gpuMat.release();
}

extern "C" void GuassFitGpuHcT(Mat matImage, MPoint *point, double maxError, double minError, int yRange, int Colonce)
{

}

